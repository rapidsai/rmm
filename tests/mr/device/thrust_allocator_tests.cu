/*
 * Copyright (c) 2019, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <gtest/gtest.h>
#include <rmm/device_vector.hpp>
#include <rmm/mr/device/thrust_allocator_adaptor.hpp>
#include "mr_test.hpp"

namespace rmm {
namespace test {
namespace {

struct allocator_test : public mr_test {
};

TEST_P(allocator_test, first)
{
  rmm::device_vector<int> ints(100, 1);
  EXPECT_EQ(100, thrust::reduce(ints.begin(), ints.end()));
}

INSTANTIATE_TEST_CASE_P(ThrustAllocatorTests,
                        allocator_test,
                        ::testing::Values(mr_factory{"CUDA", &make_cuda},
                                          mr_factory{"Managed", &make_managed},
                                          mr_factory{"Pool", &make_pool},
                                          mr_factory{"Binning", &make_binning}),
                        [](auto const& info) { return info.param.name; });
}  // namespace
}  // namespace test
}  // namespace rmm
