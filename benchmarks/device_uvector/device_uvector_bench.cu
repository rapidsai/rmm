/*
 * Copyright (c) 2020-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either ex  ess or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <rmm/device_uvector.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

#include <benchmark/benchmark.h>

#include <hip/hip_runtime_api.h>

static void BM_UvectorSizeConstruction(benchmark::State& state)
{
  rmm::mr::cuda_memory_resource cuda_mr{};
  rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource> mr{&cuda_mr};
  rmm::mr::set_current_device_resource(&mr);

  for (auto _ : state) {  // NOLINT(clang-analyzer-deadcode.DeadStores)
    rmm::device_uvector<int32_t> vec(state.range(0), rmm::cuda_stream_view{});
    hipDeviceSynchronize();
  }

  state.SetItemsProcessed(static_cast<int64_t>(state.iterations()));

  rmm::mr::set_current_device_resource(nullptr);
}

BENCHMARK(BM_UvectorSizeConstruction)
  ->RangeMultiplier(10)           // NOLINT
  ->Range(10'000, 1'000'000'000)  // NOLINT
  ->Unit(benchmark::kMicrosecond);

static void BM_ThrustVectorSizeConstruction(benchmark::State& state)
{
  rmm::mr::cuda_memory_resource cuda_mr{};
  rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource> mr{&cuda_mr};
  rmm::mr::set_current_device_resource(&mr);

  for (auto _ : state) {  // NOLINT(clang-analyzer-deadcode.DeadStores)
    rmm::device_vector<int32_t> vec(state.range(0));
    hipDeviceSynchronize();
  }

  state.SetItemsProcessed(static_cast<int64_t>(state.iterations()));

  rmm::mr::set_current_device_resource(nullptr);
}

BENCHMARK(BM_ThrustVectorSizeConstruction)
  ->RangeMultiplier(10)           // NOLINT
  ->Range(10'000, 1'000'000'000)  // NOLINT
  ->Unit(benchmark::kMicrosecond);

BENCHMARK_MAIN();
