/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either ex  ess or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <benchmark/benchmark.h>

#include <hip/hip_runtime_api.h>
#include <rmm/device_uvector.hpp>
#include <rmm/device_vector.hpp>
#include <rmm/mr/device/cuda_memory_resource.hpp>
#include <rmm/mr/device/per_device_resource.hpp>
#include <rmm/mr/device/pool_memory_resource.hpp>

static void BM_UvectorSizeConstruction(benchmark::State& state)
{
  rmm::mr::cuda_memory_resource cuda_mr{};
  rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource> mr{&cuda_mr};
  rmm::mr::set_current_device_resource(&mr);

  for (auto _ : state) {
    rmm::device_uvector<int32_t> vec(state.range(0), rmm::cuda_stream_view{});
    hipDeviceSynchronize();
  }

  state.SetItemsProcessed(state.iterations());

  rmm::mr::set_current_device_resource(nullptr);
}
BENCHMARK(BM_UvectorSizeConstruction)
  ->RangeMultiplier(10)
  ->Range(10'000, 1'000'000'000)
  ->Unit(benchmark::kMicrosecond);

static void BM_ThrustVectorSizeConstruction(benchmark::State& state)
{
  rmm::mr::cuda_memory_resource cuda_mr{};
  rmm::mr::pool_memory_resource<rmm::mr::cuda_memory_resource> mr{&cuda_mr};
  rmm::mr::set_current_device_resource(&mr);

  for (auto _ : state) {
    rmm::device_vector<int32_t> vec(state.range(0));
    hipDeviceSynchronize();
  }

  state.SetItemsProcessed(state.iterations());

  rmm::mr::set_current_device_resource(nullptr);
}

BENCHMARK(BM_ThrustVectorSizeConstruction)
  ->RangeMultiplier(10)
  ->Range(10'000, 1'000'000'000)
  ->Unit(benchmark::kMicrosecond);

BENCHMARK_MAIN();
